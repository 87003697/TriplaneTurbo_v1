#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <limits>
#include <ATen/ATen.h>
#include <iostream>
#include <tuple>
#include <functional>
#include <hip/hip_vector_types.h> // For float2, float3, float4

// <<< Add helper function needed by the new kernel >>>
namespace {
    __device__ inline float4 transformPoint4x4(const float3& p, const float* matrix) {
        float4 result;
        result.x = p.x * matrix[0] + p.y * matrix[4] + p.z * matrix[8] + matrix[12];
        result.y = p.x * matrix[1] + p.y * matrix[5] + p.z * matrix[9] + matrix[13];
        result.z = p.x * matrix[2] + p.y * matrix[6] + p.z * matrix[10] + matrix[14];
        result.w = p.x * matrix[3] + p.y * matrix[7] + p.z * matrix[11] + matrix[15];
        return result;
    }
}

// <<< Define the new Step 2 verification kernel >>>
__global__ void preprocessStep2Kernel(
    int P,
    const float* means3D,
    const float* viewmatrix,
    const float* projmatrix,
    const int W, const int H,
    float* intermediate_depths, // Still passed, but not written to in this step
    float2* intermediate_xy,      // Still passed, but not written to in this step
    float* out_depth
)
{
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= P) return;

    // Calculate p_view_z and p_proj (as before)
    float3 p_orig = {means3D[idx*3+0], means3D[idx*3+1], means3D[idx*3+2]};
    float4 p_view_h = transformPoint4x4(p_orig, viewmatrix);
    float p_view_z = p_view_h.z;
    float4 p_proj_h = transformPoint4x4(p_orig, projmatrix);
    float w = (abs(p_proj_h.w) > 1e-5) ? p_proj_h.w : 1e-5;
    float3 p_proj = {(p_proj_h.x / w + 1.f) * W / 2.f, (p_proj_h.y / w + 1.f) * H / 2.f, w};

    if (idx == 0 && blockIdx.x == 0) {
        // Keep Step 3.1 checks
        printf("[Step 3.1 Debug] idx=0: out_depth pointer = %p\n", out_depth);
        if (out_depth != nullptr) { 
             printf("[Step 3.1 Debug] idx=0: Initial out_depth[0] = %f\n", out_depth[0]);
        } else {
             printf("[Step 3.1 Debug] idx=0: out_depth pointer is NULL!\n");
        }
        // Keep Step 2 Debug print
        printf("[Step 2 Debug] idx=0: p_view.z = %f, p_proj.x = %f, p_proj.y = %f\n", p_view_z, p_proj.x, p_proj.y);
        
        // <<< Add Step 4.1 validation >>>
        int px_0 = static_cast<int>(roundf(p_proj.x - 0.5f));
        int py_0 = static_cast<int>(roundf(p_proj.y - 0.5f));
        bool in_bounds_0 = (px_0 >= 0 && px_0 < W && py_0 >= 0 && py_0 < H);
        printf("[Step 4.1 Debug] idx=0: p_proj=(%.2f, %.2f) -> px=%d, py=%d. In Bounds: %s\n",
               p_proj.x, p_proj.y, px_0, py_0, in_bounds_0 ? "YES" : "NO");
        // <<< End Step 4.1 >>>
    }

    // <<< Remove intermediate buffer writes for this step >>>
    /*
    bool valid = (p_view_z < -0.01f);
    if(valid)
    {
        intermediate_depths[idx] = p_view_z;
        intermediate_xy[idx] = make_float2(p_proj.x, p_proj.y);
    }
    else
    {
        intermediate_depths[idx] = std::numeric_limits<float>::infinity();
        intermediate_xy[idx] = make_float2(-1.f, -1.f); 
    }
    */
}

// Helper function to create a resize lambda
std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
	auto lambda = [&t](size_t N) {
		// Check if the tensor's current size (in bytes) is sufficient
		if (t.numel() * t.element_size() < N) {
			// If not, resize the tensor.
			// Note: This assumes N is the desired size in *bytes*.
			// We need to calculate the number of elements based on the tensor's dtype.
			size_t num_elements = (N + t.element_size() - 1) / t.element_size(); // Calculate elements needed
			t.resize_({(long long)num_elements}); // Resize to the required number of elements
		}
		// Return the raw data pointer as char*
		return reinterpret_cast<char*>(t.data_ptr());
	};
	return lambda; // Return the created lambda function object
}

// RasterizeGaussiansCenterDepthCUDA - Modified for Step 2 direct kernel launch
std::tuple<torch::Tensor, torch::Tensor>
RasterizeGaussiansCenterDepthCUDA(
	const torch::Tensor& means3D,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
    const float tan_fovx,
    const float tan_fovy,
    const int image_height,
    const int image_width,
    const float scale_modifier, // Unused now
    const float kernel_size,    // Unused now
    const bool prefiltered,    // Unused now
    const bool debug            // Unused now, but kept in signature
)
{
    const auto options = means3D.options();
    const int P = means3D.size(0);
    const int W = image_width;
    const int H = image_height;
    const auto device = means3D.device();

    // Create final output tensors (will remain mostly unchanged)
    auto out_opacity = torch::zeros({H, W}, options);
	auto out_depth = torch::full({H, W}, std::numeric_limits<float>::infinity(), options);

	if (P == 0) {
        return std::make_tuple(out_opacity, out_depth);
	}

    // Get pointers to inputs
    const float* means_ptr = means3D.contiguous().data_ptr<float>();
    const float* view_ptr = viewmatrix.contiguous().data_ptr<float>();
    const float* proj_ptr = projmatrix.contiguous().data_ptr<float>();

    // <<< Create and get pointers for intermediate buffers >>>
    auto intermediate_depths_tensor = torch::empty({(long long)P}, options);
    auto intermediate_xy_tensor = torch::empty({(long long)P, 2}, options); // float2
    float* intermediate_depths_ptr = intermediate_depths_tensor.data_ptr<float>();
    float2* intermediate_xy_ptr = reinterpret_cast<float2*>(intermediate_xy_tensor.data_ptr<float>());

    // <<< Launch the new Step 2 Kernel >>>
    const int threads = 128; 
    const dim3 blocks((P + threads - 1) / threads);
    preprocessStep2Kernel<<<blocks, threads>>>(
        P,
        means_ptr,
        view_ptr,
        proj_ptr,
        W, H,
        intermediate_depths_ptr,
        intermediate_xy_ptr,
        out_depth.data_ptr<float>()
    );

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error after preprocessStep2Kernel launch: %s\\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); 
    err = hipGetLastError(); // Check error again after sync
    if (err != hipSuccess) printf("CUDA Error after kernel sync: %s\\n", hipGetErrorString(err));

    // <<< Return the (unmodified) final output tensors >>>
    return std::make_tuple(out_opacity, out_depth);
} 