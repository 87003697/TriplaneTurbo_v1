#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <limits>
#include <ATen/ATen.h>
#include <iostream>
#include <tuple>
#include <functional>
#include <hip/hip_vector_types.h> // For float2, float3, float4

// <<< Add atomicMinFloat helper function >>>
__device__ inline float atomicMinFloat(float* addr, float value) {
    float old = *addr;
    while (value < old) {
        unsigned int old_int = __float_as_uint(old);
        unsigned int assumed_int = old_int; 
        unsigned int returned_int = atomicCAS((unsigned int*)addr, assumed_int, __float_as_uint(value));
        if (returned_int == assumed_int) {
            break; 
        }
        old = __uint_as_float(returned_int); 
    }
    return old;
}

// <<< Add helper function needed by the new kernel >>>
namespace {
    __device__ inline float4 transformPoint4x4(const float3& p, const float* matrix) {
        float4 result;
        result.x = p.x * matrix[0] + p.y * matrix[4] + p.z * matrix[8] + matrix[12];
        result.y = p.x * matrix[1] + p.y * matrix[5] + p.z * matrix[9] + matrix[13];
        result.z = p.x * matrix[2] + p.y * matrix[6] + p.z * matrix[10] + matrix[14];
        result.w = p.x * matrix[3] + p.y * matrix[7] + p.z * matrix[11] + matrix[15];
        return result;
    }
}

// <<< Kernel for Step 4-3b: Full atomic write >>>
__global__ void fullAtomicMinKernel(
    int P,
    const float* means3D,
    const float* viewmatrix,
    const float* projmatrix,
    const int W, const int H,
    float* out_depth // Only need final output depth pointer
)
{
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= P) return;

    // Calculate p_view_z and p_proj 
    float3 p_orig = {means3D[idx*3+0], means3D[idx*3+1], means3D[idx*3+2]};
    float4 p_view_h = transformPoint4x4(p_orig, viewmatrix);
    float p_view_z = p_view_h.z;
    float4 p_proj_h = transformPoint4x4(p_orig, projmatrix);
    float w = (abs(p_proj_h.w) > 1e-5) ? p_proj_h.w : 1e-5;
    float3 p_proj = {(p_proj_h.x / w + 1.f) * W / 2.f, (p_proj_h.y / w + 1.f) * H / 2.f, w};

    // Calculate pixel coordinates 
    int px = static_cast<int>(roundf(p_proj.x - 0.5f));
    int py = static_cast<int>(roundf(p_proj.y - 0.5f));

    // Bounds and validity checks 
    if (px >= 0 && px < W && py >= 0 && py < H) {
        if (p_view_z < -0.01f && isfinite(p_view_z)) {
            int pix_id = py * W + px;
            // --- Perform atomicMinFloat --- 
            atomicMinFloat(&out_depth[pix_id], p_view_z);
        }
    }
}

// Helper function to create a resize lambda
std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
	auto lambda = [&t](size_t N) {
		// Check if the tensor's current size (in bytes) is sufficient
		if (t.numel() * t.element_size() < N) {
			// If not, resize the tensor.
			// Note: This assumes N is the desired size in *bytes*.
			// We need to calculate the number of elements based on the tensor's dtype.
			size_t num_elements = (N + t.element_size() - 1) / t.element_size(); // Calculate elements needed
			t.resize_({(long long)num_elements}); // Resize to the required number of elements
		}
		// Return the raw data pointer as char*
		return reinterpret_cast<char*>(t.data_ptr());
	};
	return lambda; // Return the created lambda function object
}

// <<< Modify function to return TWO tensors and launch the correct kernel >>>
std::tuple<torch::Tensor, torch::Tensor>
RasterizeGaussiansCenterDepthCUDA(
    const torch::Tensor& means3D,
    const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
    const float tan_fovx,
    const float tan_fovy,
    const int image_height,
    const int image_width,
    const float scale_modifier,
    const float kernel_size,
    const bool prefiltered,
    const bool debug)
{
    const auto options = means3D.options();
    const int P = means3D.size(0);
    const int W = image_width;
    const int H = image_height;
    const auto device = means3D.device();

    auto out_opacity = torch::zeros({H, W}, options); // Opacity remains 0 for now
	auto out_depth = torch::full({H, W}, std::numeric_limits<float>::infinity(), options);

	if (P == 0) {
        return std::make_tuple(out_opacity, out_depth);
	}

    // Get pointers
    const float* means_ptr = means3D.contiguous().data_ptr<float>();
    const float* view_ptr = viewmatrix.contiguous().data_ptr<float>();
    const float* proj_ptr = projmatrix.contiguous().data_ptr<float>();
	float* depth_ptr = out_depth.data_ptr<float>();

    // <<< Launch the fullAtomicMinKernel >>>
    const int threads = 128; 
    const dim3 blocks((P + threads - 1) / threads);
    fullAtomicMinKernel<<<blocks, threads>>>(
        P,
        means_ptr,
        view_ptr,
        proj_ptr,
        W, H,
        depth_ptr
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error after fullAtomicMinKernel launch: %s\\n", hipGetErrorString(err));
    }
    // hipDeviceSynchronize(); // Optional for timing, not needed for correctness here

    // <<< Return TWO tensors >>>
    return std::make_tuple(out_opacity, out_depth);
} 