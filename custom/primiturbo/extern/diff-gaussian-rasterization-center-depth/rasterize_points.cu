#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <limits>
#include <ATen/ATen.h>
#include <iostream>
#include <tuple>
#include <functional>
#include <hip/hip_vector_types.h> // For float2, float3, float4

// <<< Add helper function needed by the new kernel >>>
namespace {
    __device__ inline float4 transformPoint4x4(const float3& p, const float* matrix) {
        float4 result;
        result.x = p.x * matrix[0] + p.y * matrix[4] + p.z * matrix[8] + matrix[12];
        result.y = p.x * matrix[1] + p.y * matrix[5] + p.z * matrix[9] + matrix[13];
        result.z = p.x * matrix[2] + p.y * matrix[6] + p.z * matrix[10] + matrix[14];
        result.w = p.x * matrix[3] + p.y * matrix[7] + p.z * matrix[11] + matrix[15];
        return result;
    }
}

// <<< Define the new Step 2 verification kernel >>>
__global__ void preprocessStep2Kernel(
    int P,
    const float* means3D,
    const float* viewmatrix,
    const float* projmatrix,
    const int W, const int H,
    float* intermediate_depths, // Output buffer 1
    float2* intermediate_xy      // Output buffer 2
)
{
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= P) return;

    float3 p_orig = {means3D[idx*3+0], means3D[idx*3+1], means3D[idx*3+2]};

    // Transform point by view matrix
    float4 p_view_h = transformPoint4x4(p_orig, viewmatrix);
    float p_view_z = p_view_h.z; // View space depth

    // Project to screen space
    float4 p_proj_h = transformPoint4x4(p_orig, projmatrix);
    float w = (abs(p_proj_h.w) > 1e-5) ? p_proj_h.w : 1e-5;
    float3 p_proj = {(p_proj_h.x / w + 1.f) * W / 2.f, (p_proj_h.y / w + 1.f) * H / 2.f, w};

    // <<< Step 2 Add printf >>>
    if (idx == 0 && blockIdx.x == 0)
    {
        printf("[Step 2 Debug] idx=0: p_view.z = %f, p_proj.x = %f, p_proj.y = %f\n", p_view_z, p_proj.x, p_proj.y);
    }

    // Check validity (example: point in front of camera)
    bool valid = (p_view_z < -0.01f);

    if(valid)
    {
        // Write to intermediate buffers
        intermediate_depths[idx] = p_view_z;
        intermediate_xy[idx] = make_float2(p_proj.x, p_proj.y);
    }
    else
    {
        intermediate_depths[idx] = std::numeric_limits<float>::infinity();
        intermediate_xy[idx] = make_float2(-1.f, -1.f); // Mark as invalid
    }
}

// Helper function to create a resize lambda
std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
	auto lambda = [&t](size_t N) {
		// Check if the tensor's current size (in bytes) is sufficient
		if (t.numel() * t.element_size() < N) {
			// If not, resize the tensor.
			// Note: This assumes N is the desired size in *bytes*.
			// We need to calculate the number of elements based on the tensor's dtype.
			size_t num_elements = (N + t.element_size() - 1) / t.element_size(); // Calculate elements needed
			t.resize_({(long long)num_elements}); // Resize to the required number of elements
		}
		// Return the raw data pointer as char*
		return reinterpret_cast<char*>(t.data_ptr());
	};
	return lambda; // Return the created lambda function object
}

// RasterizeGaussiansCenterDepthCUDA - Modified for Step 2 direct kernel launch
std::tuple<torch::Tensor, torch::Tensor>
RasterizeGaussiansCenterDepthCUDA(
	const torch::Tensor& means3D,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
    const float tan_fovx,
    const float tan_fovy,
    const int image_height,
    const int image_width,
    const float scale_modifier, // Unused now
    const float kernel_size,    // Unused now
    const bool prefiltered,    // Unused now
    const bool debug            // Unused now, but kept in signature
)
{
    const auto options = means3D.options();
    const int P = means3D.size(0);
    const int W = image_width;
    const int H = image_height;
    const auto device = means3D.device();

    // Create final output tensors (will remain mostly unchanged)
    auto out_opacity = torch::zeros({H, W}, options);
	auto out_depth = torch::full({H, W}, std::numeric_limits<float>::infinity(), options);

	if (P == 0) {
        return std::make_tuple(out_opacity, out_depth);
	}

    // Get pointers to inputs
    const float* means_ptr = means3D.contiguous().data_ptr<float>();
    const float* view_ptr = viewmatrix.contiguous().data_ptr<float>();
    const float* proj_ptr = projmatrix.contiguous().data_ptr<float>();

    // <<< Create and get pointers for intermediate buffers >>>
    auto intermediate_depths_tensor = torch::empty({(long long)P}, options);
    auto intermediate_xy_tensor = torch::empty({(long long)P, 2}, options); // float2
    float* intermediate_depths_ptr = intermediate_depths_tensor.data_ptr<float>();
    float2* intermediate_xy_ptr = reinterpret_cast<float2*>(intermediate_xy_tensor.data_ptr<float>());

    // <<< Launch the new Step 2 Kernel >>>
    const int threads = 128; 
    const dim3 blocks((P + threads - 1) / threads);
    preprocessStep2Kernel<<<blocks, threads>>>(
        P,
        means_ptr,
        view_ptr,
        proj_ptr,
        W, H,
        intermediate_depths_ptr,
        intermediate_xy_ptr
    );

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error after preprocessStep2Kernel launch: %s\\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); // Synchronize to ensure printf finishes

    // <<< Return the (unmodified) final output tensors >>>
    return std::make_tuple(out_opacity, out_depth);
} 