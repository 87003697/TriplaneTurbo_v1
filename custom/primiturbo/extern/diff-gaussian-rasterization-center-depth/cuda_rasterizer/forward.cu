#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <limits>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
template<bool INTE = false>
__device__ bool computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, float kernel_size, const float* cov3D, const float* viewmatrix, 
							float* cov2D, float* camera_plane, float3* output_normal, float2* ray_plane, float& coef, float* invraycov3Ds = nullptr)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	float txtz = t.x / t.z;
	float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;
	txtz = t.x / t.z;
	tytz = t.y / t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// output[0] = { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
	cov2D[0] = float(cov[0][0] + kernel_size);
	cov2D[1] = float(cov[0][1]);
	cov2D[2] = float(cov[1][1] + kernel_size);
	const float det_0 = max(1e-6, cov[0][0] * cov[1][1] - cov[0][1] * cov[0][1]);
	const float det_1 = max(1e-6, (cov[0][0] + kernel_size) * (cov[1][1] + kernel_size) - cov[0][1] * cov[0][1]);
	coef = sqrt(det_0 / (det_1+1e-6) + 1e-6);
	if (det_0 <= 1e-6 || det_1 <= 1e-6){
		coef = 0.0f;
	}

	// glm::mat3 testm = glm::mat3{
	// 	1,2,3,
	// 	4,5,6,
	// 	7,8,9,
	// };
	// glm::vec3 testv = {1,1,1};
	// glm::vec3 resultm = testm * testv;
	// printf("%f %f %f\n", resultm[0], resultm[1],resultm[2]); 12.000000 15.000000 18.000000

	glm::mat3 Vrk_eigen_vector;
	glm::vec3 Vrk_eigen_value;
	int D = glm_modification::findEigenvaluesSymReal(Vrk,Vrk_eigen_value,Vrk_eigen_vector);

	unsigned int min_id = Vrk_eigen_value[0]>Vrk_eigen_value[1]? (Vrk_eigen_value[1]>Vrk_eigen_value[2]?2:1):(Vrk_eigen_value[0]>Vrk_eigen_value[2]?2:0);

	glm::mat3 Vrk_inv;
	bool well_conditioned = Vrk_eigen_value[min_id]>0.00000001;
	glm::vec3 eigenvector_min;
	if(well_conditioned)
	{
		glm::mat3 diag = glm::mat3( 1/Vrk_eigen_value[0], 0, 0,
									0, 1/Vrk_eigen_value[1], 0,
									0, 0, 1/Vrk_eigen_value[2] );
		Vrk_inv = Vrk_eigen_vector * diag * glm::transpose(Vrk_eigen_vector);
	}
	else
	{
		eigenvector_min = Vrk_eigen_vector[min_id];
		Vrk_inv = glm::outerProduct(eigenvector_min,eigenvector_min);
	}
	
	glm::mat3 cov_cam_inv = glm::transpose(W) * Vrk_inv * W;
	glm::vec3 uvh = {txtz, tytz, 1};
	glm::vec3 uvh_m = cov_cam_inv * uvh;
	glm::vec3 uvh_mn = glm::normalize(uvh_m);

	if(isnan(uvh_mn.x)|| D==0)
	{
		for(int ch = 0; ch < 6; ch++)
			camera_plane[ch] = 0;
		*output_normal = {0,0,0};
		*ray_plane = {0,0};
	}
	else
	{
		float u2 = txtz * txtz;
		float v2 = tytz * tytz;
		float uv = txtz * tytz;

		float l = sqrt(t.x*t.x+t.y*t.y+t.z*t.z);
		glm::mat3 nJ = glm::mat3(
			1 / t.z, 0.0f, -(t.x) / (t.z * t.z),
			0.0f, 1 / t.z, -(t.y) / (t.z * t.z),
			t.x/l, t.y/l, t.z/l);

		glm::mat3 nJ_inv = glm::mat3(
			v2 + 1,	-uv, 		0,
			-uv,	u2 + 1,		0,
			-txtz,	-tytz,		0
		);

		if constexpr (INTE)
		{
			glm::mat3 inv_cov_ray;
			if(well_conditioned)
			{
				float ltz = u2+v2+1;
				glm::mat3 nJ_inv_full = t.z/(u2+v2+1) * \
										glm::mat3(
											v2 + 1,	-uv, 		txtz/l*ltz,
											-uv,	u2 + 1,		tytz/l*ltz,
											-txtz,	-tytz,		1/l*ltz);
				glm::mat3 T2 = W * glm::transpose(nJ_inv_full);
				inv_cov_ray = glm::transpose(T2) * Vrk_inv * T2;
			}
			else
			{
				glm::mat3 T2 = W * nJ;
				glm::mat3 cov_ray = glm::transpose(T2) * Vrk_inv * T2;
				glm::mat3 cov_eigen_vector;
				glm::vec3 cov_eigen_value;
				glm_modification::findEigenvaluesSymReal(cov_ray,cov_eigen_value,cov_eigen_vector);
				unsigned int min_id = cov_eigen_value[0]>cov_eigen_value[1]? (cov_eigen_value[1]>cov_eigen_value[2]?2:1):(cov_eigen_value[0]>cov_eigen_value[2]?2:0);
				float lambda1 = cov_eigen_value[(min_id+1)%3];
				float lambda2 = cov_eigen_value[(min_id+2)%3];
				float lambda3 = cov_eigen_value[min_id];
				glm::mat3 new_cov_eigen_vector = glm::mat3();
				new_cov_eigen_vector[0] = cov_eigen_vector[(min_id+1)%3];
				new_cov_eigen_vector[1] = cov_eigen_vector[(min_id+2)%3];
				new_cov_eigen_vector[2] = cov_eigen_vector[min_id];
				glm::vec3 r3 = glm::vec3(new_cov_eigen_vector[0][2],new_cov_eigen_vector[1][2],new_cov_eigen_vector[2][2]);

				glm::mat3 cov2d = glm::mat3(
					1/lambda1,0,-r3[0]/r3[2]/lambda1,
					0,1/lambda2,-r3[1]/r3[2]/lambda2,
					-r3[0]/r3[2]/lambda1,-r3[1]/r3[2]/lambda2,0
				);
				glm::mat3 inv_cov_ray = new_cov_eigen_vector * cov2d * glm::transpose(new_cov_eigen_vector);
			}
			glm::mat3 scale = glm::mat3(1/focal_x,0,0,
										0, 1/focal_y,0,
										0,0,1);
			inv_cov_ray = scale * inv_cov_ray * scale;
			invraycov3Ds[0] = inv_cov_ray[0][0];
			invraycov3Ds[1] = inv_cov_ray[0][1];
			invraycov3Ds[2] = inv_cov_ray[0][2];
			invraycov3Ds[3] = inv_cov_ray[1][1];
			invraycov3Ds[4] = inv_cov_ray[1][2];
			invraycov3Ds[5] = inv_cov_ray[2][2];
		}


		float vbn = glm::dot(uvh_mn, uvh);
		float factor_normal = l / (u2+v2+1);
		glm::vec3 plane = nJ_inv * (uvh_mn/max(vbn,0.0000001f));
		float nl = u2+v2+1;
		glm::vec2 camera_plane_x = {(-(v2 + 1)*t.z+plane[0]*t.x)/nl/focal_x, (uv*t.z+plane[1]*t.x)/nl/focal_y};
		glm::vec2 camera_plane_y = {(uv*t.z+plane[0]*t.y)/nl/focal_x, (-(u2 + 1)*t.z+plane[1]*t.y)/nl/focal_y};
		glm::vec2 camera_plane_z = {(t.x+plane[0]*t.z)/nl/focal_x, (t.y+plane[1]*t.z)/nl/focal_y};

		*ray_plane = {plane[0]*l/nl/focal_x, plane[1]*l/nl/focal_y};

		camera_plane[0] = camera_plane_x.x;
		camera_plane[1] = camera_plane_x.y;
		camera_plane[2] = camera_plane_y.x;
		camera_plane[3] = camera_plane_y.y;
		camera_plane[4] = camera_plane_z.x;
		camera_plane[5] = camera_plane_z.y;


		glm::vec3 ray_normal_vector = {-plane[0]*factor_normal, -plane[1]*factor_normal, -1};
		glm::vec3 cam_normal_vector = nJ * ray_normal_vector;
		glm::vec3 normal_vector = glm::normalize(cam_normal_vector);

		*output_normal = {normal_vector.x, normal_vector.y, normal_vector.z};

	}
	return well_conditioned;
}


// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C, bool INTE = false>
    __global__ void preprocessCUDA(
        int P,
        const float* means3D,
        const float* viewmatrix,
        const float* projmatrix,
        const float tan_fovx, const float tan_fovy,
        const int W, const int H,
        float2* points_xy_image, // float2*
        float* depths,          // float*
        float* out_opacity,     // float*
        float* out_depth,       // float*
        bool debug)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;
	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;
	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	float cov2D[3];
	float ceof;
	bool condition = computeCov2D<INTE>(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, kernel_size, cov3D, viewmatrix, cov2D, camera_planes + idx * 6, normals + idx, ray_planes + idx, ceof, invraycov3Ds + idx * 6);
	if constexpr (INTE)
	{
		conditions[idx] = condition;
	}
	ts[idx] = sqrt(p_view.x*p_view.x+p_view.y*p_view.y+p_view.z*p_view.z);
	const float3 cov = {cov2D[0], cov2D[1], cov2D[2]};
	
	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	view_points[idx] = p_view;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] * ceof};
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);

	// <<< Step 2 Add printf >>>
	if (idx == 0 && blockIdx.x == 0)
	{
		printf("[Step 2 Debug] idx=0: p_view.z = %f, p_proj.x = %f, p_proj.y = %f\n", p_view.z, p_proj.x, p_proj.y);
	}

	// Check validity
	bool valid = (p_view.z < -0.01f);

	if (valid) {
		// <<< Step 2 Write to intermediate buffers (UNCOMMENTED) >>>
		depths[idx] = p_view.z;
		points_xy_image[idx] = make_float2(p_proj.x, p_proj.y);
		
		// <<< Step 2 Ensure final output writes are COMMENTED OUT >>>
		/*
		// Example:
		// int px = ...; int py = ...;
		// if(...) { int pix_id = ...; out_depth[pix_id] = ...; out_opacity[pix_id] = ...; }
		*/
	} else {
		// Mark invalid points
		depths[idx] = std::numeric_limits<float>::infinity();
		points_xy_image[idx] = make_float2(-1.f, -1.f);
		// <<< Step 2 Ensure final output writes are COMMENTED OUT >>>
		/*
		// Example:
		// int px = ...; int py = ...;
		// if(...) { int pix_id = ...; out_depth[pix_id] = ...; out_opacity[pix_id] = ...; }
		*/
	}
	// <<< Ensure Step 3 specific debug writes/prints are REMOVED >>>
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS, bool COORD, bool DEPTH, bool NORMAL>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float* __restrict__ view_points,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float* __restrict__ ts,
	const float* __restrict__ camera_planes,
	const float2* __restrict__ ray_planes,
	const float3* __restrict__ normals,
	const float4* __restrict__ conic_opacity,
	const float focal_x, 
	const float focal_y,
	float* __restrict__ out_alpha,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	float* __restrict__ out_coord,
	float* __restrict__ out_mcoord,
	float* __restrict__ out_normal,
	float* __restrict__ out_depth,
	float* __restrict__ out_mdepth,
	float* __restrict__ accum_coord,
	float* __restrict__ accum_depth,
	float* __restrict__ normal_length
	)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };
	float2 pixnf = {(pixf.x-W/2.f)/focal_x,(pixf.y-H/2.f)/focal_y};
	float ln = sqrt(pixnf.x*pixnf.x+pixnf.y*pixnf.y+1);

	constexpr bool GEO = DEPTH || COORD || NORMAL;

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float collected_feature[BLOCK_SIZE * CHANNELS];
	__shared__ float collected_camera_plane[BLOCK_SIZE * 6];
	__shared__ float collected_mean3d[BLOCK_SIZE * 3];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_ts[BLOCK_SIZE];
	__shared__ float2 collected_ray_planes[BLOCK_SIZE];
	__shared__ float3 collected_normals[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	uint32_t max_contributor = -1;
	float C[CHANNELS] = { 0 };
	float weight = 0;
	float Coord[3] = { 0 };
	float mCoord[3] = { 0 };
	float Depth = 0;
	float mDepth = 0;
	float Normal[3] = {0};
	float last_depth = 0;
	float last_weight = 0;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			for(int ch = 0; ch < CHANNELS; ch++)
				collected_feature[ch * BLOCK_SIZE + block.thread_rank()] = features[coll_id * CHANNELS + ch];
			if constexpr (COORD)
			{
				for(int ch = 0; ch < 6; ch++)
					collected_camera_plane[ch * BLOCK_SIZE + block.thread_rank()] = camera_planes[coll_id * 6 + ch];
				for(int ch = 0; ch < 3; ch++)
					collected_mean3d[ch * BLOCK_SIZE + block.thread_rank()] = view_points[coll_id * 3 + ch];
			}
			if constexpr (DEPTH)
			{
				collected_ts[block.thread_rank()] = ts[coll_id];
				collected_ray_planes[block.thread_rank()] = ray_planes[coll_id];
			}
			if constexpr (NORMAL)
			{
				collected_normals[block.thread_rank()] = normals[coll_id];
			}
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f){
				continue;
			}
				

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			const float aT = alpha * T;
			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += collected_feature[j + BLOCK_SIZE * ch] * aT;

			bool before_median = T > 0.5;
			if constexpr (COORD)
			{
				float2 camera_plane0 = {collected_camera_plane[j], collected_camera_plane[j + BLOCK_SIZE]};
				float2 camera_plane1 = {collected_camera_plane[j + BLOCK_SIZE * 2], collected_camera_plane[j + BLOCK_SIZE * 3]};
				float2 camera_plane2 = {collected_camera_plane[j + BLOCK_SIZE * 4], collected_camera_plane[j + BLOCK_SIZE * 5]};
				float coord[3] = {collected_mean3d[j] + camera_plane0.x * d.x + camera_plane0.y * d.y,
									collected_mean3d[j + BLOCK_SIZE] + camera_plane1.x * d.x + camera_plane1.y * d.y,
									collected_mean3d[j + BLOCK_SIZE * 2] + camera_plane2.x * d.x + camera_plane2.y * d.y};
				for(int ch = 0; ch < 3; ch++)
					Coord[ch] += coord[ch] * aT;
				if (before_median){
					for(int ch = 0; ch < 3; ch++)
						mCoord[ch] = coord[ch];
				}
			}
			if constexpr (DEPTH)
			{
				float t_center = collected_ts[j];
				float2 ray_plane = collected_ray_planes[j];
				float t = t_center + (ray_plane.x * d.x + ray_plane.y * d.y);
				// float depth = t/ln;
				Depth += t * aT;
				if (before_median) mDepth = t;
			}

			if constexpr (NORMAL)
			{
				Normal[0] += collected_normals[j].x * aT;
				Normal[1] += collected_normals[j].y * aT;
				Normal[2] += collected_normals[j].z * aT;
			}
			
			if constexpr (GEO)
			{
				if (before_median)
					max_contributor = contributor;
			}

			
			weight += aT;
			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		n_contrib[pix_id] = last_contributor;
		n_contrib[pix_id + H * W] = max_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		out_alpha[pix_id] = weight; //1 - T;

		if constexpr (COORD)
		{
			if(last_contributor)
			{
				for (int ch = 0; ch < 3; ch++)
				{
					out_coord[ch * H * W + pix_id] = Coord[ch] / weight;
				}
			}
			else
			{
				for (int ch = 0; ch < 3; ch++)
					out_coord[ch * H * W + pix_id] = 0;
			}
			for (int ch = 0; ch < 3; ch++)
			{
				accum_coord[ch * H * W + pix_id] = Coord[ch];
				out_mcoord[ch * H * W + pix_id] = mCoord[ch];
			}
		}

		if constexpr (DEPTH)
		{
			float depth_ln = Depth/ln;
			accum_depth[pix_id] = depth_ln;
			if(last_contributor)
			{
				out_depth[pix_id] = depth_ln/weight;
			}
			else
			{
				out_depth[pix_id] = 0;
			}
			out_mdepth[pix_id] = mDepth/ln;
		}

		if constexpr (NORMAL)
		{
			if(last_contributor)
			{
				float len_normal = sqrt(Normal[0]*Normal[0]+Normal[1]*Normal[1]+Normal[2]*Normal[2]);
				normal_length[pix_id] = len_normal;
				len_normal = max(len_normal, NORMALIZE_EPS);
				for (int ch = 0; ch < 3; ch++)
					out_normal[ch * H * W + pix_id] = Normal[ch]/len_normal;
			}
			else
			{
				normal_length[pix_id] = 1;
				for (int ch = 0; ch < 3; ch++)
					out_normal[ch * H * W + pix_id] = 0;
			}
		}
	}
}

// the Bool inputs can be replaced by an enumeration variable for different functions.
void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float* view_points,
	const float2* means2D,
	const float* colors,
	const float* ts,
	const float* camera_planes,
	const float2* ray_planes,
	const float3* normals,
	const float4* conic_opacity,
	const float focal_x, float focal_y,
	float* out_alpha,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	float* out_coord,
	float* out_mcoord,
	float* out_normal,
	float* out_depth,
	float* out_mdepth,
	float* accum_coord,
	float* accum_depth,
	float* normal_length,
	bool require_coord,
	bool require_depth)
{
#define RENDER_CUDA_CALL(template_coord, template_depth, template_normal) \
renderCUDA<NUM_CHANNELS, template_coord, template_depth, template_normal> <<<grid, block>>> ( \
	ranges, point_list, W, H, view_points, means2D, colors, ts, camera_planes, ray_planes, \
	normals, conic_opacity, focal_x, focal_y, out_alpha, n_contrib, bg_color, out_color, \
	out_coord, out_mcoord, out_normal, out_depth, out_mdepth, \
	accum_coord, accum_depth, normal_length)

	if (require_coord && require_depth)
		RENDER_CUDA_CALL(true, true, true);
	else if (require_coord && !require_depth)
		RENDER_CUDA_CALL(true, false, true);
	else if(!require_coord && require_depth)
		RENDER_CUDA_CALL(false, true, true);
	else
		RENDER_CUDA_CALL(false, false, false);
		
#undef RENDER_CUDA_CALL
}

namespace FORWARD
{
    // Function definition for preprocess - Match corrected forward.h signature
    void preprocess(
        int P, int D, int M,
	const float* means3D,
        const float* scales,
	const float scale_modifier,
        const float* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
        const float* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const float kernel_size,
	int* radii,
	float3* view_points,
	float* depths,
	float* camera_planes,
	float2* ray_planes,
	float* ts,
	float3* normals,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered,
        float2* points_xy_image,
        float* out_opacity,
        float* out_depth,
        bool debug,
	bool integrate,
	float* invraycov3Ds,
	bool* condition)
{
        const int threads = 128;
        const dim3 blocks((P + threads - 1) / threads);

        // Launch preprocessCUDA kernel with the simplified signature
        preprocessCUDA<<<blocks, threads>>>(
            P,
			means3D,
			viewmatrix, 
			projmatrix,
			tan_fovx, tan_fovy,
            W, H,
            points_xy_image,
			depths,
            out_opacity,
            out_depth,
            debug
        );
    }

    // Kernel definition for preprocessCUDA - **FORCE SIMPLIFIED SIGNATURE (4th attempt)**
    __global__ void preprocessCUDA(
        // Parameters MUST exactly match the 13 arguments in the launch call
        int P,
        const float* means3D,
        const float* viewmatrix,
        const float* projmatrix,
        const float tan_fovx, const float tan_fovy,
        const int W, const int H,
        float2* points_xy_image, // float2*
        float* depths,          // float*
        float* out_opacity,     // float*
        float* out_depth,       // float*
        bool debug)
    {
        // ... Keep the kernel implementation the same ...
        auto idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= P) return;

        float3 p_orig = {means3D[idx*3+0], means3D[idx*3+1], means3D[idx*3+2]};

        float4 p_view_h = transformPoint4x4(p_orig, viewmatrix);
        float p_view_z = p_view_h.z; 

        float4 p_proj_h = transformPoint4x4(p_orig, projmatrix);
        float w = (abs(p_proj_h.w) > 1e-5) ? p_proj_h.w : 1e-5;
        float3 p_proj = {(p_proj_h.x / w + 1.f) * W / 2.f, (p_proj_h.y / w + 1.f) * H / 2.f, w};

        if (idx == 0 && blockIdx.x == 0) {
            printf("[Step 2 Debug] idx=0: p_view.z = %f, p_proj.x = %f, p_proj.y = %f\n", p_view_z, p_proj.x, p_proj.y);
        }

        bool valid = (p_view_z < -0.01f);

        if (valid) {
            depths[idx] = p_view_z;
            points_xy_image[idx] = make_float2(p_proj.x, p_proj.y);
            /* // Final writes commented out */
        } else {
            depths[idx] = std::numeric_limits<float>::infinity();
            points_xy_image[idx] = make_float2(-1.f, -1.f);
            /* // Final writes commented out */
        }
    }

    // ... rest of FORWARD namespace ...
}

namespace {
    // Function to transform point by 4x4 matrix
    __device__ inline float4 transformPoint4x4(const float3& p, const float* matrix) {
        float4 result;
        result.x = p.x * matrix[0] + p.y * matrix[4] + p.z * matrix[8] + matrix[12];
        result.y = p.x * matrix[1] + p.y * matrix[5] + p.z * matrix[9] + matrix[13];
        result.z = p.x * matrix[2] + p.y * matrix[6] + p.z * matrix[10] + matrix[14];
        result.w = p.x * matrix[3] + p.y * matrix[7] + p.z * matrix[11] + matrix[15];
        return result;
    }
}
